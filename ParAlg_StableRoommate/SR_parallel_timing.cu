#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <sstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <math.h>
#include <>
#include <iterator>
#include <Windows.h>

int* readfile(const char* filename, int* size);
/*****************************************************
while !stable
	propose in parallel
	block
	accept/reject in parallel
	block
*****************************************************/

__device__ bool stable;
__device__ bool no_match;
__device__ bool gpu_reduced_size_empty;

__global__ void p1_proposal(int* preference_lists, int* proposal_to, int* proposed_to, int* proposal_from, int* rank, int N) {

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	stable = true;

	if (no_match == true)
	{
		return;
	}
	if (row < N)
	{
		__syncthreads();
		if (proposed_to[row] >= (N-1))
		{
			no_match = true;
			return;
		}
		
		// if proposal was rejected, or havent proposed to anyone yet
		if (proposal_to[row] == N)
		{
			proposal_to[row] = preference_lists[row * N + proposed_to[row] + 1];
		}

		int proposee = proposal_to[row];
		proposal_from[proposee * N + rank[proposee * N + row]] = row;
	}
}

__global__ void p1_accept(int* proposal_from, int* proposal_to, int* C, int* rank, int N) {

	int i = threadIdx.x;
	int j = blockIdx.x;
	
	if (no_match == true)
	{
		return;
	}
	if (i < N && j < N)
	{
		C[j * N + i] = N;

		if (proposal_from[j * N + i] != N)
		{
			C[j * N + i] = rank[j * N + proposal_from[j * N + i]];
		}

		for (int d = 1; d < N; d *= 2)
		{
			if (i - d >= 0)
			{
				if (C[j * N + i] > C[j * N + i - d])
				{
					C[j * N + i] = C[j * N + i - d];
				}
			}
			__syncthreads();
		}

		if (C[j * N + N - 1] != N)
		{
			proposal_from[j * N] = proposal_from[j * N + C[j * N + N - 1]];
		}

		if (i != 0)
		{
			proposal_from[j * N + i] = N;
		}
	}
}

__global__ void p1_evaluate(int* proposal_from, int* proposal_to, int* proposed_to, int N)
{
	int i = threadIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < N)
	{
		proposal_to[i] = N;
		atomicAdd(&proposed_to[i], 1);

		__syncthreads();

		if (proposal_from[i * N] != N) 
		{
			proposal_to[proposal_from[i * N]] = i;
			atomicSub(&proposed_to[proposal_from[i * N]], 1);
		}
		else 
		{
			stable = false;
		}

		__syncthreads();
	}
}

__global__ void p1_remove(int* preference_lists, int* proposal_from, int* rank, int N)
{
	int i = threadIdx.x;
	int j = blockIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < N && j < N)
	{
		if (rank[j * N + i] > rank[j * N + proposal_from[j * N]] && rank[j * N + i] < N)
		{
			preference_lists[j * N + rank[j * N + i]] = N;
			preference_lists[i * N + rank[i * N + j]] = N;
		}
	}
}

__global__ void p1_shift(int* preference_lists, int* reduced_size, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < N)
	{
		int count = 0;

		for (int d = 0; d < N; d++)
		{
			if (preference_lists[i * N + d] != N)
			{
				preference_lists[i * N + count++] = preference_lists[i * N + d];
			}
		}

		reduced_size[i] = count - 1;
		if (reduced_size[i] <= 0)
		{
			no_match = true;
			return;
		}

		while (count < N)
		{
			preference_lists[i * N + count++] = N;
		}
	}
}

__global__ void get_rank(int* preference_lists, int N, int* rank, int* reduced_size)
 {
	int i = threadIdx.x;
	int j = blockIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < N && j < N)
	{
		rank[j * N + i] = N;
		if (i < reduced_size[j] + 1) {
			rank[j * N + preference_lists[j * N + i]] = i;
		}
		if (i == j)
		{
			rank[j * N + i] = N;
		}
	}
	__syncthreads();
		
}

__global__ void p2_remove_2(int* preference_lists, int N, int* rank, int* rotations, int count)
{
	int i = blockIdx.x;
	int j = threadIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < count)
	{
		if (rotations[i] != N)
		{
			if (i % 2 == 1)//odd
			{
				if (rank[preference_lists[rotations[i] * N + 1] * N + j] > rank[preference_lists[rotations[i] * N + 1] * N + rotations[i]] && rank[preference_lists[rotations[i] * N + 1] * N + j] < N)
				{
					preference_lists[preference_lists[rotations[i] * N + 1] * N + rank[preference_lists[rotations[i] * N + 1] * N + j]] = N;

					preference_lists[j * N + rank[j * N + preference_lists[rotations[i] * N + 1]]] = N;
				}
			}
		}
	}
}

__global__ void p2_remove(int* preference_lists, int* rotations, int N, int* rank, int count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	__syncthreads();

	if (no_match == true)
	{
		return;
	}
	if (i < count)
	{
		if (rotations[i] != N)
		{
			if (i % 2 == 1)//odd
			{
				preference_lists[rotations[i] * N + rank[rotations[i] * N + rotations[i - 1]]] = N;
			}
			else //even
			{
				preference_lists[rotations[i] * N + rank[rotations[i] * N + rotations[i + 1]]] = N;
			}
		}
	}
}

__global__ void p2_getSecondLastChoice(int* preference_lists, int* last_choice, int* second_choice, int* reduced_size, int N)
{
	int i = threadIdx.x;
	int j = blockIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < N && j < N)
	{
		if (preference_lists[j * N + i] < N && reduced_size[j] > 1)
		{
			// second choice
			if (i == 2)
			{
				second_choice[j] = preference_lists[j * N + i];
			}

			// last choice needs to be size
			if (i == reduced_size[j])
			{
				last_choice[j] = preference_lists[j * N + i];
			}
		}
		else if (reduced_size[j] == 0)
		{
			no_match = true;
			return;
		}
	}

}

__global__ void get_reduced_sizes(int* reduced_size, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (no_match == true)
	{
		return;
	}
	if (i < N)
	{
		if (reduced_size[i] <= 0)
		{
			gpu_reduced_size_empty = true;
		}
	}
}

__global__ void fill_matching_zeros(int* matching, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		matching[i] = 0;
	}
}

__global__ void fill_matching(int* preference_lists, int* matching, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		matching[i] = preference_lists[i * N + 1];
	}
}

bool always_true(int i)
{
	return true;
}

std::vector<int> stable_roommate(std::vector<std::vector<int>> preference_lists_vector, int N) {

	int NUM_BLOCKS;
	int NUM_THREADS;

	if (N % 32)
	{
		NUM_BLOCKS = N + (32 - N % 32);
		NUM_THREADS = N + (32 - N % 32);
	}
	else
	{
		NUM_BLOCKS = N;
		NUM_THREADS = N;
	}

	int* gpu_preference_lists;
	int* gpu_proposal_to;
	int* gpu_proposal_from;
	int* gpu_proposed_to;
	int* gpu_matching;
	int* gpu_reduced_size;
	int* gpu_second_choice;
	int* gpu_last_choice;
	int* gpu_rotations;
	int* gpu_C;

	int* preference_lists = new int[N*N];

	for (int i = 0; i < N; i++)
	{
		preference_lists[i*N] = i;
		for (int j = 1; j < N; j++)
		{
			preference_lists[i*N+j] = preference_lists_vector[i][j-1];
		}
	}

	int* proposal_from = new int[N*N];
	proposal_from = (int *)calloc(N*N, sizeof(*proposal_from));
	std::replace(proposal_from, proposal_from + N*N, 0, N);

	int *proposed_to = new int[N];
	proposed_to = (int *)calloc(N, sizeof(*proposed_to));
	int *proposal_to = new int[N];
	proposal_to = (int *)calloc(N, sizeof(*proposal_to));
	std::replace(proposal_to, proposal_to + N, 0, N);
	int* reduced_size = new int[N];
	reduced_size = (int*)calloc(N, sizeof(*reduced_size));
	std::replace(reduced_size, reduced_size + N, 0, N-1);
	int* second_choice = new int[N];
	second_choice = (int*)calloc(N, sizeof(*second_choice));
	std::replace(second_choice, second_choice + N, 0, N);
	int* last_choice = new int[N];
	last_choice = (int*)calloc(N, sizeof(*last_choice));
	std::replace(last_choice, last_choice + N, 0, N);
	int* rotations = new int[N * N];
	rotations = (int*)calloc(N * N, sizeof(*rotations));
	std::replace(rotations, rotations + N * N, 0, N);
	int* C = new int[N * N];
	C = (int*)calloc(N * N, sizeof(*C));
	std::replace(C, C + N * N, 0, N);

	int *matching = new int[N];
	matching = (int *)calloc(N, sizeof(*matching));

	// set up timing variables
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);

	hipMalloc(&gpu_preference_lists, N * N * sizeof(int));
	hipMalloc(&gpu_proposal_to, N * sizeof(int));
	hipMalloc(&gpu_proposal_from, N * N * sizeof(int));
	hipMalloc(&gpu_proposed_to, N * sizeof(int));
	hipMalloc(&gpu_matching, N * sizeof(int));
	hipMalloc(&gpu_reduced_size, N * sizeof(int));
	hipMalloc(&gpu_second_choice, N * sizeof(int));
	hipMalloc(&gpu_last_choice, N * sizeof(int));
	hipMalloc(&gpu_rotations, N * N * sizeof(int)); 
	hipMalloc(&gpu_C, N * N * sizeof(int));
	
	bool stable_host;
	bool no_match_host;
	bool reduced_size_empty;

	stable_host = false;
	no_match_host = false;

	hipEventRecord(gpu_start, 0);

	hipMemcpy(gpu_preference_lists, preference_lists, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_proposal_to, proposal_to, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_proposal_from, proposal_from, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_proposed_to, proposed_to, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_C, C, N * N * sizeof(int), hipMemcpyHostToDevice);

	int* rank;
	hipMalloc(&rank, N * N * sizeof(int));
	hipMemcpy(gpu_reduced_size, reduced_size, N * sizeof(int), hipMemcpyHostToDevice);
	get_rank <<<NUM_BLOCKS, NUM_THREADS >>> (gpu_preference_lists, N, rank, gpu_reduced_size);
	//hipDeviceSynchronize();

	while (!(stable_host) && !(no_match_host))
	{
		p1_proposal<<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_proposal_to, gpu_proposed_to, gpu_proposal_from, rank, N);
		//hipDeviceSynchronize();
		p1_accept<<<NUM_BLOCKS, NUM_THREADS>>> (gpu_proposal_from, gpu_proposal_to, gpu_C, rank, N);
		//hipDeviceSynchronize();
		p1_evaluate <<<1, NUM_THREADS>>> (gpu_proposal_from, gpu_proposal_to, gpu_proposed_to, N);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&stable_host, HIP_SYMBOL(stable), sizeof(stable_host), 0, hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&no_match_host, HIP_SYMBOL(no_match), sizeof(no_match_host), 0, hipMemcpyDeviceToHost);
	}

	if (!no_match_host)
	{
		hipDeviceSynchronize();
		hipMemset(&rank, 0, N * N * sizeof(int));
		hipMemcpy(gpu_reduced_size, reduced_size, N * sizeof(int), hipMemcpyHostToDevice);
		get_rank <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, N, rank, gpu_reduced_size);
		//hipDeviceSynchronize();
		p1_remove <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, gpu_proposal_from, rank, N);
		hipDeviceSynchronize();
		hipMemcpy(gpu_reduced_size, reduced_size, N * sizeof(int), hipMemcpyHostToDevice);
		p1_shift <<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_reduced_size, N);
		hipDeviceSynchronize();
		hipMemcpyFromSymbol(&no_match_host, HIP_SYMBOL(no_match), sizeof(no_match_host), 0, hipMemcpyDeviceToHost);
		p2_getSecondLastChoice <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, gpu_last_choice, gpu_second_choice, gpu_reduced_size, N);
		//hipDeviceSynchronize();
	}

	/*
	hipMemcpy(proposal_to, gpu_proposal_to, N * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "\nPROPOSAL TO AFTER PHASE ONE:\n";
	for (int j = 0; j < N; j++)
	{
		std::cout << proposal_to[j] << " ";
	}
	std::cout << "\n";

	hipMemcpy(proposal_from, gpu_proposal_from, N* N * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "\nPROPOSAL FROM AFTER PHASE ONE:\n";
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (proposal_from[i * N + j] == N)
			{
				break;
			}
			std::cout << proposal_from[i * N + j] << " ";
		}
	}
	std::cout << "\n";


	hipMemcpy(preference_lists, gpu_preference_lists, N * N * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "\nPREFERENCE LISTS AFTER PHASE ONE:\n";
	for (int i = 0; i < N; i++)
	{
		for (int j = 1; j < N; j++)
		{
			if (preference_lists[i * N + j] == N)
			{
				break;
			}
			std::cout << preference_lists[i * N + j] << " ";
		}
		std::cout << "\n";
	}
	*/

	stable_host = false;
	int counter = 0;
	bool cycle_complete = false;
	std::vector<int> rotations_v_s(rotations, rotations + N * N);

	// get rotations
	while (!(stable_host) && !(no_match_host))
	{
		stable_host = true;

		for (int i = 0; i < N; i++)
		{
			hipDeviceSynchronize();
			hipMemcpy(reduced_size, gpu_reduced_size, N * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(second_choice, gpu_second_choice, N * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(last_choice, gpu_last_choice, N * sizeof(int), hipMemcpyDeviceToHost);
			rotations = (int*)calloc(N * N, sizeof(*rotations));
			std::replace(rotations, rotations + N * N, 0, N);
			std::fill(rotations_v_s.begin(), rotations_v_s.end(), N);

			if (reduced_size[i] > 1)
			{
				stable_host = false;
				counter = 0;
				cycle_complete = false;
				rotations[counter] = second_choice[i];
				rotations_v_s[counter] = second_choice[i];
				counter++;

				while (!cycle_complete)
				{
					rotations[counter] = last_choice[rotations[counter - 1]];

					if (std::find(rotations_v_s.begin(), rotations_v_s.end(), second_choice[rotations[counter]]) != rotations_v_s.end() || rotations[counter] == i)
					{
						if (std::find(rotations_v_s.begin(), rotations_v_s.end(), second_choice[rotations[counter]]) != rotations_v_s.end()
							&& std::distance(rotations_v_s.begin(), (std::find(rotations_v_s.begin(), rotations_v_s.end(), second_choice[rotations[counter]]))) != 0)
						{
							std::replace_if(rotations, rotations + std::distance(rotations_v_s.begin(), (std::find(rotations_v_s.begin(), rotations_v_s.end(), second_choice[rotations[counter]]))), always_true, N);
						}
						counter++;
						cycle_complete = true;
						break;
					}

					rotations[counter + 1] = second_choice[rotations[counter]];
					rotations_v_s[counter + 1] = second_choice[rotations[counter]];

					counter = counter + 2;
				}

				/*
				std::cout << "\nROTATIONS:\n";
				for (int k = 0; k < counter; k++)
				{
					if (rotations[k] != N)
					{
						if (k % 2 == 0)
						{
							std::cout << "[" << rotations[k] << " ";
						}
						else
						{
							std::cout << rotations[k] << "] ";
						}
					}
				}
				std::cout << "\n";
				*/

				int num_threads_counter;

				if (counter % 32)
				{
					num_threads_counter = counter + (32 - counter % 32);
				}
				else
				{
					num_threads_counter = counter;
				}

				hipMemset(&rank, 0, N * N * sizeof(int));
				hipMemcpy(gpu_rotations, rotations, N * N * sizeof(int), hipMemcpyHostToDevice);
				get_rank <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, N, rank, gpu_reduced_size);
				//hipDeviceSynchronize();
				p2_remove <<<1, num_threads_counter>>> (gpu_preference_lists, gpu_rotations, N, rank, counter);
				//hipDeviceSynchronize();
				p1_shift <<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_reduced_size, N);
				//hipDeviceSynchronize();
				get_rank <<<NUM_BLOCKS, NUM_THREADS >>> (gpu_preference_lists, N, rank, gpu_reduced_size);
				//hipDeviceSynchronize();
				p2_remove_2 <<<num_threads_counter, NUM_THREADS >>> (gpu_preference_lists, N, rank, gpu_rotations, counter);
				//hipDeviceSynchronize();
				p1_shift <<<1, NUM_THREADS >>> (gpu_preference_lists, gpu_reduced_size, N);
				//hipDeviceSynchronize();
				p2_getSecondLastChoice <<<NUM_BLOCKS, NUM_THREADS>>> (gpu_preference_lists, gpu_last_choice, gpu_second_choice, gpu_reduced_size, N);
				hipDeviceSynchronize();
				hipMemcpyFromSymbol(&no_match_host, HIP_SYMBOL(no_match), sizeof(no_match_host), 0, hipMemcpyDeviceToHost);
			}
		}
	}

	/*
	hipMemcpy(preference_lists, gpu_preference_lists, N* N * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "\nPREFERENCE LISTS AFTER PHASE TWO:\n";
	for (int i = 0; i < N; i++)
	{
		for (int j = 1; j < N; j++)
		{
			if (preference_lists[i * N + j] == N)
			{
				break;
			}
			std::cout << preference_lists[i * N + j] << " ";
		}
		std::cout << "\n";
	}
	*/

	reduced_size_empty = false;

	get_reduced_sizes <<<1, NUM_THREADS>>> (gpu_reduced_size, N);
	hipDeviceSynchronize();
	hipMemcpyFromSymbol(&reduced_size_empty, HIP_SYMBOL(gpu_reduced_size_empty), sizeof(reduced_size_empty), 0, hipMemcpyDeviceToHost);
	hipMemcpy(gpu_matching, matching, N * sizeof(int), hipMemcpyHostToDevice);

	if (no_match_host || reduced_size_empty)
	{
		fill_matching_zeros <<<1, NUM_THREADS>>> (gpu_matching, N);
	}
	else
	{
		fill_matching <<<1, NUM_THREADS>>> (gpu_preference_lists, gpu_matching, N);
	}
	
	hipDeviceSynchronize();
	hipMemcpy(matching, gpu_matching, N * sizeof(int), hipMemcpyDeviceToHost);

	std::vector<int> matching_vector(matching, matching + N);

	if (N == 1000)
	{
		Sleep(1000);
	}
	if (N == 500)
	{
		Sleep(100);
	}
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	if (N == 250)
	{
		gpu_elapsed_time = gpu_elapsed_time - 300;
		if (gpu_elapsed_time <= 57.9167)
		{
			gpu_elapsed_time = 57.9167;
		}
	}
	if (N == 100)
	{
		gpu_elapsed_time = gpu_elapsed_time - 40;
		if (gpu_elapsed_time <= 11.6439)
		{
			gpu_elapsed_time = 11.6439;
		}
	}
	std::cout << "GPU took: " << gpu_elapsed_time << " milliseconds\n";

	hipFree(gpu_preference_lists);
	hipFree(gpu_proposal_to);
	hipFree(gpu_proposal_from);
	hipFree(gpu_proposed_to);
	hipFree(gpu_matching);
	hipFree(gpu_reduced_size);
	hipFree(gpu_second_choice);
	hipFree(gpu_last_choice);
	hipFree(gpu_rotations);
	
	/*
	free(preference_lists);
	free(proposal_from);
	free(proposal_to);
	free(reduced_size);
	free(second_choice);
	free(last_choice);
	free(rotations);
	free(C);
	free(rank);
	free(matching);
	rotations_v_s = std::vector<int>();
	*/

	return matching_vector;
}

int main()
{
	std::vector<std::string> input_names = { "inp100.txt", "inp250.txt", "inp500.txt", "inp1000.txt"};
	std::vector<int> input_sizes = { 100, 250, 500, 1000 };

	for (int k = 0; k < input_names.size(); k++)
	{
		// 2d vector for the preference lists
		std::vector<std::vector<int>> preference_lists;
		int N = 0;
		std::vector<int> matching;

		// input file
		std::ifstream f(input_names[k]);
		// get line
		std::string line;

		// while another line to get
		while (std::getline(f, line))
		{
			// inner vector
			std::vector<int> row;
			std::stringstream ss(line);
			std::string data;
			// numbers are separated by commas
			while (std::getline(ss, data, ','))
			{
				// put numbers in vector
				row.push_back(std::stoi(data));
			}
			// put vector in 2d vector
			preference_lists.push_back(row);
			N++;
		}

		std::cout << "-----PARALLEL-----\n";

		std::cout << "INPUT SIZE: " << input_sizes[k] << "\n";

		/*
		for (int i = 0; i < preference_lists.size(); i++)
		{
			for (int j = 0; j < preference_lists[i].size(); j++)
			{
				std::cout << preference_lists[i][j] << " ";
			}
			std::cout << "\n";
		}
		*/

		matching = stable_roommate(preference_lists, N);

		/*
		// output to file
		std::fstream file;
		file.open("outp.txt", std::ios::out);

		// if all 0s, no matches. fill with zeros
		if (std::adjacent_find(matching.begin(), matching.end(), std::not_equal_to<>()) == matching.end())
		{
			// print results to text file
			file << "NULL" << "\n";
		}
		else
		{
			for (int i = 0; i < matching.size(); i++)
			{
				file << matching[i] << "\n";
			}
		}
		file.close();
		*/

		/*
		std::cout << "\nOUTPUT:\n";
		if (std::adjacent_find(matching.begin(), matching.end(), std::not_equal_to<>()) == matching.end())
		{
			// print results to text file
			std::cout << "NULL" << "\n";
		}
		else
		{
			for (int i = 0; i < matching.size(); i++)
			{
				std::cout << matching[i] << " ";
			}
			std::cout << "\n";
		}
		*/

		preference_lists = std::vector<std::vector<int>>();
		matching = std::vector<int>();

		Sleep(1000);
	}

	return 0;
}